#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>

#include <functional>
#include <random>
#include <iostream>

// read from data
#include "data/config.h.in"

// std::uniform_int_distribution
template<class DATA_TYPE, template<class> class DISTRIBUTE>
std::function<DATA_TYPE(const std::vector<int>&)> get_rand_data_gen(
    DATA_TYPE lowwer_bound,
    DATA_TYPE upper_bound
) {
    std::random_device rd;
    std::mt19937 gen(rd());
    DISTRIBUTE<DATA_TYPE> dist(lowwer_bound, upper_bound);
    return [dist, gen] (const std::vector<int>& in) mutable {return dist(gen);};
}



// result value check of cuda runtime
#define CHECK(call) check(call, __LINE__, __FILE__)

inline bool check(hipError_t e, int iLine, const char *szFile)
{
    if (e != hipSuccess)
    {
        std::cout << "CUDA runtime API error " << hipGetErrorName(e) << " at line " << iLine << " in file " << szFile << std::endl;
        return false;
    }
    return true;
}

#if 0
constexpr int N = 7;
constexpr int D = 120;
constexpr int IH = 64;
constexpr int IW = 120;
constexpr int C = 128;
constexpr int OH = 80;
constexpr int OW = 160;
constexpr int P = 1606542;
#endif

extern "C"
int get_config(char *config) {
  int ret = -1;
  if (config == nullptr)
    return ret;

  char c1 = *config, c2 = *(config + 1);
  switch (c1) {
  case 'N':
    ret = N;
    break;
  case 'D':
    ret = D;
    break;
  case 'I':
    ret = (c2 == 'H') ? IH : (c2 == 'W') ? IW : -1;
    break;
  case 'O':
    ret = (c2 == 'H') ? OH : (c2 == 'W') ? OW : -1;
    break;
  case 'C':
    ret = C;
    break;
  case 'P':
    ret = P;
    break;
  default:
    ret = -1;
    break;
  }
  return ret;
}

void read_file(const char *filename, size_t element_sz, size_t size, void *buffer) {
  FILE *fp = fopen(filename, "rb");
  if (fp == nullptr) {
    printf("fopen error: %s\n", filename);
    return;
  }
  size_t num = fread(buffer, element_sz, size, fp);
  if (num != size) {
    printf("read error: %s\n", filename);
  };
  fclose(fp);
}

extern "C"
void tensor_init(int *ranks_depth,
                 int *ranks_feat,
                 int *ranks_bev,
                 int *interval_starts,
                 int *interval_lengths,
                 int8_t *ranks_bev_mask,
                 int *interval_starts_e,
                 int *interval_lengths_e,
                 int *interval_vids_e,
                 int *interval_starts_x,
                 int *interval_lengths_x,
                 int *interval_vids_x,
                 int *n_intervals_x) {

  read_file("data/ranks_depth.bin", sizeof(float), 4000000, ranks_depth);
  read_file("data/ranks_feat.bin", sizeof(float), 4000000, ranks_feat);
  read_file("data/ranks_bev.bin", sizeof(float), 4000000, ranks_bev);
  read_file("data/interval_starts.bin", sizeof(float), 50000, interval_starts);
  read_file("data/interval_lengths.bin", sizeof(float), 50000, interval_lengths);

  for (int i = 0; i < 4000000; i++) {
    *(int*)&ranks_bev[i] = (int)*(float*)&ranks_bev[i];
    *(int*)&ranks_depth[i] = (int)*(float*)&ranks_depth[i];
    *(int*)&ranks_feat[i] = (int)*(float*)&ranks_feat[i];
    int idx = ranks_bev[i];
    if (idx != -1 && ranks_bev_mask[idx] == 0)
      ranks_bev_mask[idx] = 1;
  }
  for (int i = 0; i < 50000; i++) {
    *(int*)&interval_starts[i] = (int)*(float*)&interval_starts[i];
    *(int*)&interval_lengths[i] = (int)*(float*)&interval_lengths[i];
  }
  int j = 0;
  for (int i = 0; i < 50000; i++) {
    if (i >= (OH * OW)) {
        interval_vids_e[i] = -1;
        interval_starts_e[i] = -1;
        interval_lengths_e[i] = -1;
    } else {
        interval_vids_e[i] = i;
        if (ranks_bev_mask[i] == 0) {
            interval_starts_e[i] = 0;
            interval_lengths_e[i] = 0;
        } else {
            interval_starts_e[i] = interval_starts[j];
            interval_lengths_e[i] = interval_lengths[j];
            j++;
        }
    }
  }

  int k = 0;
  for (int i = 0; i < 50000; i++) {
    int len = interval_lengths[i];
    if (len != -1) {
      if (len <= MIL) {
        interval_starts_x[k] = interval_starts[i];
        interval_lengths_x[k] = interval_lengths[i];
        interval_vids_x[k] = ranks_bev[interval_starts[i]];
        k++;
      } else {
        int ext = (len + MIL - 1) / MIL;
        for  (int e = 0; e < ext; e++) {
          interval_lengths_x[k] = (e < (ext - 1)) ? MIL : (len - MIL * e);
          interval_starts_x[k] = interval_starts[i] + MIL * e;
          interval_vids_x[k] = ranks_bev[interval_starts[i]];
          k++;
        }
      }
    }
  }
  *n_intervals_x = k;
}


extern "C" void tensor_NDHW_to_NHWD(int *ranks_depth, size_t num, int N, int D, int H, int W);
void tensor_NDHW_to_NHWD(int *ranks_depth, size_t num, int N, int D, int H, int W) {
#pragma omp parallel for
  for (int i = 0; i < num; i++) {
    int idx = ranks_depth[i];
      if (idx != -1) {

        int n = idx / (D*H*W);
        int d = (idx - n*D*H*W)/(H*W);
        int h = (idx - n*D*H*W - d*H*W)/W;
        int w = (idx - n*D*H*W - d*H*W - h*W);

        int new_idx = n*H*W*D + h*W*D + w*D + d;
        ranks_depth[i] = new_idx;
      }
  }
}


template<typename TensorType, typename AccType, const int TC, const int TN>
__global__ void bev_pool_flatmap_kernel(
    int C, int N, const TensorType *__restrict__ depth,
    const TensorType *__restrict__ feat, const int *__restrict__ ranks_depth,
    const int *__restrict__ ranks_feat, const int *__restrict__ ranks_bev,
    const int *__restrict__ interval_starts,
    const int *__restrict__ interval_lengths, TensorType *__restrict__ out) {

  int tc_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int tn_idx = blockIdx.y * blockDim.y + threadIdx.y;

  for (int tc = 0; tc < TC; tc++) {
    int c_idx = tc_idx * TC + tc;
    if (c_idx >= C) continue;

    int b_idx_last = -1;
    int b_idx = 0;
    TensorType psum = 0;

    for (int tn = 0; tn < TN; tn++) {
      int n_idx = tn_idx * TN + tn;
      if (n_idx >= N) continue;
      b_idx = ranks_bev[n_idx];

      TensorType d = depth[ranks_depth[n_idx]];
      TensorType f = feat[ranks_feat[n_idx]*C + c_idx];

      if (b_idx == b_idx_last) {
        psum += d*f;
      } else {
        if (b_idx_last != -1)
          atomicAdd(&out[b_idx_last*C + c_idx], psum);
        b_idx_last = b_idx;
        psum = d * f;
      }
    }

    if (b_idx_last != -1)
      atomicAdd(&out[b_idx_last*C + c_idx], psum);
  }
}

extern "C"
void bev_pool_flatmap(int C, int n_intervals, const float *depth, const float *feat,
                      const int *ranks_depth, const int *ranks_feat,
                      const int *ranks_bev, const int *interval_starts,
                      const int *interval_lengths, float *out) {

  constexpr int local_TC = 1;
  constexpr int local_TN = 90;
  constexpr int local_BC = 96;
  constexpr int local_BN = 2;
  dim3 gridSize((C + local_TC * local_BC - 1)/(local_TC * local_BC), (P + local_TN * local_BN - 1)/(local_TN * local_BN));
  dim3 blockSize(local_BC, local_BN);

  hipMemset(out, 0, OH*OW*C*sizeof(float));
  bev_pool_flatmap_kernel<float, float, local_TC, local_TN><<<gridSize, blockSize>>>(
      C, P, depth, feat, ranks_depth, ranks_feat, ranks_bev,
      interval_starts, interval_lengths, out);
}

template<typename InType, typename AccType, typename OutType, const int TC, const int TN, const int IHW=1, const int OHW=1, bool FEAT_CL=true>
__global__ void bev_pool_kernel(
    int C, int n_intervals,
    const InType *__restrict__ depth,
    const InType *__restrict__ feat,
    const int *__restrict__ ranks_depth,
    const int *__restrict__ ranks_feat,
    const int *__restrict__ interval_starts,
    const int *__restrict__ interval_lengths,
    OutType *__restrict__ out) {

  int tc_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int tn_idx = blockIdx.y * blockDim.y + threadIdx.y;

#pragma unroll
  for (int tn = 0; tn < TN; tn++) {
    AccType psum[TC];
    int n_idx = tn_idx * TN + tn;
    if (n_idx >= n_intervals) break;

    int interval_start = __ldg(&interval_starts[n_idx]);
    int interval_length = __ldg(&interval_lengths[n_idx]);

    if (interval_start == -1) break;

    for (int tc = 0; tc < TC; tc++) {
      psum[tc] = 0;
    }

    for (int i = 0; i < interval_length; i++) {
      InType d = __ldg(&depth[ranks_depth[interval_start + i]]);
#pragma unroll
      for (int tc = 0; tc < TC; tc++) {
        int c_idx = tc_idx * TC + tc;
        if (c_idx >= C) continue;

        InType f;
        if constexpr (FEAT_CL)
          f = __ldg(&feat[ranks_feat[interval_start + i] * C + c_idx]);
        else {// NCHW
          int idx = ranks_feat[interval_start + i];
          int n = idx / IHW;
          int hw = idx % IHW;
          f = __ldg(&feat[n*C*IHW + c_idx*IHW + hw]);
        }

        if constexpr (std::is_same<InType, __half>::value && std::is_same<AccType, float>::value)
          psum[tc] = __fmaf_rn(__half2float(d), __half2float(f), psum[tc]);
        else if constexpr (std::is_same<InType, __hip_bfloat16>::value && std::is_same<AccType, float>::value)
          psum[tc] = __fmaf_rn(__bfloat162float(d), __bfloat162float(f), psum[tc]);
        else if constexpr (std::is_same<InType, float>::value && std::is_same<AccType, float>::value)
          psum[tc] = __fmaf_rn(d, f, psum[tc]);
        else
          psum[tc] += d * f;
      }
    }

#pragma unroll
    for (int tc = 0; tc < TC; tc++) {
      int c_idx = tc_idx * TC + tc;
      if (c_idx >= C) break;
      int tid;
      if constexpr (FEAT_CL)
        tid = n_idx * C + c_idx;
      else {
        int n = n_idx / OHW;
        int hw = n_idx % OHW;
        tid = n*C*OHW + c_idx*OHW + hw;
      }
      if constexpr (std::is_same<OutType, __half>::value && std::is_same<AccType, float>::value)
        out[tid] = __float2half(psum[tc]);
      else if constexpr (std::is_same<OutType, __hip_bfloat16>::value && std::is_same<AccType, float>::value)
        out[tid] = __float2bfloat16(psum[tc]);
      else
        out[tid] = psum[tc];
    }
  }
}

extern "C"
void bev_pool_float_float_float(int c, int n_intervals,
                                const float *depth,
                                const float *feat,
                                const int *ranks_depth,
                                const int *ranks_feat,
                                const int *interval_starts,
                                const int *interval_lengths,
                                float *out) {
  dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
  dim3 blockSize(BC, BN);
  bev_pool_kernel<float, float, float, TC, TN><<<gridSize, blockSize>>>(
      c, n_intervals, depth, feat, ranks_depth, ranks_feat,
      interval_starts, interval_lengths, out);
}


extern "C"
void bev_pool_float_float_float_nchw(int c, int n_intervals,
                                     const float *depth,
                                     const float *feat,
                                     const int *ranks_depth,
                                     const int *ranks_feat,
                                     const int *interval_starts,
                                     const int *interval_lengths,
                                     float *out) {
  dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
  dim3 blockSize(BC, BN);
  bev_pool_kernel<float, float, float, TC, TN, IH*IW, OH*OW, false><<<gridSize, blockSize>>>(
      c, n_intervals, depth, feat, ranks_depth, ranks_feat,
      interval_starts, interval_lengths, out);
}

extern "C"
void bev_pool_half_float_half(int c, int n_intervals,
                              const __half *depth,
                              const __half *feat,
                              const int *ranks_depth,
                              const int *ranks_feat,
                              const int *interval_starts,
                              const int *interval_lengths,
                              __half *out) {
  dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
  dim3 blockSize(BC, BN);
  bev_pool_kernel<__half, float, __half, TC, TN><<<gridSize, blockSize>>>(
      c, n_intervals, depth, feat, ranks_depth, ranks_feat,
      interval_starts, interval_lengths, out);
}

extern "C"
void bev_pool_half_float_float(int c, int n_intervals,
                                const __half *depth,
                                const __half *feat,
                                const int *ranks_depth,
                                const int *ranks_feat,
                                const int *interval_starts,
                                const int *interval_lengths,
                                float *out) {
  dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
  dim3 blockSize(BC, BN);
  bev_pool_kernel<__half, float, float, TC, TN><<<gridSize, blockSize>>>(
      c, n_intervals, depth, feat, ranks_depth, ranks_feat,
      interval_starts, interval_lengths, out);
}


extern "C"
void bev_pool_bf16_float_bf16(int c, int n_intervals,
                              const __hip_bfloat16 *depth,
                              const __hip_bfloat16 *feat,
                              const int *ranks_depth,
                              const int *ranks_feat,
                              const int *interval_starts,
                              const int *interval_lengths,
                              __hip_bfloat16 *out) {
  dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
  dim3 blockSize(BC, BN);
  bev_pool_kernel<__hip_bfloat16, float, __hip_bfloat16, TC, TN><<<gridSize, blockSize>>>(
      c, n_intervals, depth, feat, ranks_depth, ranks_feat,
      interval_starts, interval_lengths, out);
}

extern "C"
void bev_pool_bf16_bf16_bf16(int c, int n_intervals,
                             const __hip_bfloat16 *depth,
                             const __hip_bfloat16 *feat,
                             const int *ranks_depth,
                             const int *ranks_feat,
                             const int *interval_starts,
                             const int *interval_lengths,
                             __hip_bfloat16 *out) {
  dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
  dim3 blockSize(BC, BN);
  bev_pool_kernel<__hip_bfloat16, __hip_bfloat16, __hip_bfloat16, TC, TN><<<gridSize, blockSize>>>(
      c, n_intervals, depth, feat, ranks_depth, ranks_feat,
      interval_starts, interval_lengths, out);
}

extern "C"
void bev_pool_bf16_float_float(int c, int n_intervals,
                               const __hip_bfloat16 *depth,
                               const __hip_bfloat16 *feat,
                               const int *ranks_depth,
                               const int *ranks_feat,
                               const int *interval_starts,
                               const int *interval_lengths,
                               float *out) {
  dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
  dim3 blockSize(BC, BN);
  bev_pool_kernel<__hip_bfloat16, float, float, TC, TN><<<gridSize, blockSize>>>(
      c, n_intervals, depth, feat, ranks_depth, ranks_feat,
      interval_starts, interval_lengths, out);
}

extern "C"
void bev_pool_half_half_half(int c, int n_intervals,
                        const __half *depth,
                        const __half *feat,
                        const int *ranks_depth,
                        const int *ranks_feat,
                        const int *interval_starts,
                        const int *interval_lengths,
                        __half *out) {
  dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
  dim3 blockSize(BC, BN);
  bev_pool_kernel<__half, __half, __half, TC, TN><<<gridSize, blockSize>>>(
      c, n_intervals, depth, feat, ranks_depth, ranks_feat,
      interval_starts, interval_lengths, out);
}

template<typename DType, typename FType, typename OType, const int TC, const int TN>
__global__ void bev_pool_kernel_v2(
    int C, int n_intervals,
    const DType *__restrict__ depth,
    const FType *__restrict__ feat,
    const int *__restrict__ ranks_depth,
    const int *__restrict__ ranks_feat,
    const int *__restrict__ interval_starts,
    const int *__restrict__ interval_lengths,
    OType *__restrict__ out) {

  int tc_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int tn_idx = blockIdx.y * blockDim.y + threadIdx.y;

#pragma unroll
  for (int tn = 0; tn < TN; tn++) {
    float psum[TC];
    int n_idx = tn_idx * TN + tn;
    if (n_idx >= n_intervals) break;

    int interval_start = __ldg(&interval_starts[n_idx]);
    int interval_length = __ldg(&interval_lengths[n_idx]);

    if (interval_start == -1) break;

    for (int tc = 0; tc < TC; tc++) {
      psum[tc] = 0;
    }

    for (int i = 0; i < interval_length; i++) {
      float d = (float)__ldg(&depth[ranks_depth[interval_start + i]]);
#pragma unroll
      for (int tc = 0; tc < TC; tc++) {
        int c_idx = tc_idx * TC + tc;
        if (c_idx >= C) continue;

        float f = (float)__ldg(&feat[ranks_feat[interval_start + i] * C + c_idx]);
        psum[tc] = __fmaf_rn(d, f, psum[tc]);
      }
    }

#pragma unroll
    for (int tc = 0; tc < TC; tc++) {
      int c_idx = tc_idx * TC + tc;
      if (c_idx >= C) break;
      int tid;
      tid = n_idx * C + c_idx;
      out[tid] = psum[tc];
    }
  }
}

extern "C"
void bev_pool_v2_float_float_float(int c, int n_intervals,
                                   const float *depth,
                                   const float *feat,
                                   const int *ranks_depth,
                                   const int *ranks_feat,
                                   const int *interval_starts,
                                   const int *interval_lengths,
                                   float *out) {
    dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
    dim3 blockSize(BC, BN);
    bev_pool_kernel_v2<float, float, float, TC, TN><<<gridSize, blockSize>>>(
        c, n_intervals, depth, feat, ranks_depth, ranks_feat,
        interval_starts, interval_lengths, out);
}

extern "C"
void bev_pool_v2_float_half_half(int c, int n_intervals,
                                 const float *depth,
                                 const __half *feat,
                                 const int *ranks_depth,
                                 const int *ranks_feat,
                                 const int *interval_starts,
                                 const int *interval_lengths,
                                 __half *out) {
    dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
    dim3 blockSize(BC, BN);
    bev_pool_kernel_v2<float, __half, __half, TC, TN><<<gridSize, blockSize>>>(
        c, n_intervals, depth, feat, ranks_depth, ranks_feat,
        interval_starts, interval_lengths, out);
}

extern "C"
void bev_pool_v2_float_half_float(int c, int n_intervals,
                                 const float *depth,
                                 const __half *feat,
                                 const int *ranks_depth,
                                 const int *ranks_feat,
                                 const int *interval_starts,
                                 const int *interval_lengths,
                                 float *out) {
    dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
    dim3 blockSize(BC, BN);
    bev_pool_kernel_v2<float, __half, float, TC, TN><<<gridSize, blockSize>>>(
        c, n_intervals, depth, feat, ranks_depth, ranks_feat,
        interval_starts, interval_lengths, out);
}

extern "C"
void bev_pool_v2_half_half_float(int c, int n_intervals,
                                  const __half *depth,
                                  const __half *feat,
                                  const int *ranks_depth,
                                  const int *ranks_feat,
                                  const int *interval_starts,
                                  const int *interval_lengths,
                                  float *out) {
    dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
    dim3 blockSize(BC, BN);
    bev_pool_kernel_v2<__half, __half, float, TC, TN><<<gridSize, blockSize>>>(
        c, n_intervals, depth, feat, ranks_depth, ranks_feat,
        interval_starts, interval_lengths, out);
}

extern "C"
void bev_pool_v2_half_float_float(int c, int n_intervals,
                                  const __half *depth,
                                  const float *feat,
                                  const int *ranks_depth,
                                  const int *ranks_feat,
                                  const int *interval_starts,
                                  const int *interval_lengths,
                                  float *out) {
    dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
    dim3 blockSize(BC, BN);
    bev_pool_kernel_v2<__half, float, float, TC, TN><<<gridSize, blockSize>>>(
        c, n_intervals, depth, feat, ranks_depth, ranks_feat,
        interval_starts, interval_lengths, out);
}

extern "C"
void bev_pool_v2_half_float_half(int c, int n_intervals,
                                  const __half *depth,
                                  const float *feat,
                                  const int *ranks_depth,
                                  const int *ranks_feat,
                                  const int *interval_starts,
                                  const int *interval_lengths,
                                  __half *out) {
    dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
    dim3 blockSize(BC, BN);
    bev_pool_kernel_v2<__half, float, __half, TC, TN><<<gridSize, blockSize>>>(
        c, n_intervals, depth, feat, ranks_depth, ranks_feat,
        interval_starts, interval_lengths, out);
}


template<typename DType, typename FType, typename OType, const int TC, const int TN>
__global__ void bev_pool_kernel_v2_outchannelfirst(
    int C, int n_intervals,
    const DType *__restrict__ depth,
    const FType *__restrict__ feat,
    const int *__restrict__ ranks_depth,
    const int *__restrict__ ranks_feat,
    const int *__restrict__ interval_starts,
    const int *__restrict__ interval_lengths,
    OType *__restrict__ out) {

  // int tc_idx = blockIdx.x * blockDim.x + threadIdx.x;
  // int tn_idx = blockIdx.y * blockDim.y + threadIdx.y;
  int tn_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int tc_idx = blockIdx.y * blockDim.y + threadIdx.y;

#pragma unroll
  for (int tn = 0; tn < TN; tn++) {
    float psum[TC];
    int n_idx = tn_idx * TN + tn;
    if (n_idx >= n_intervals) break;

    int interval_start = __ldg(&interval_starts[n_idx]);
    int interval_length = __ldg(&interval_lengths[n_idx]);

    if (interval_start == -1) break;

    for (int tc = 0; tc < TC; tc++) {
      psum[tc] = 0;
    }

    for (int i = 0; i < interval_length; i++) {
      float d = (float)__ldg(&depth[ranks_depth[interval_start + i]]);
#pragma unroll
      for (int tc = 0; tc < TC; tc++) {
        int c_idx = tc_idx * TC + tc;
        if (c_idx >= C) continue;

        float f = (float)__ldg(&feat[ranks_feat[interval_start + i] * C + c_idx]);
        psum[tc] = __fmaf_rn(d, f, psum[tc]);
      }
    }

#pragma unroll
    for (int tc = 0; tc < TC; tc++) {
      int c_idx = tc_idx * TC + tc;
      if (c_idx >= C) break;
      int tid;
      // tid = n_idx * C + c_idx;
      tid = c_idx * OH * OW + n_idx;
      out[tid] = psum[tc];
    }
  }
}

extern "C"
void bev_pool_v2_float_float_float_outchannelfirst(int c, int n_intervals,
                                   const float *depth,
                                   const float *feat,
                                   const int *ranks_depth,
                                   const int *ranks_feat,
                                   const int *interval_starts,
                                   const int *interval_lengths,
                                   float *out) {
    // dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
    // dim3 blockSize(BC, BN);
    dim3 gridSize((n_intervals + TN * BN - 1)/(TN * BN), (c + TC * BC - 1)/(TC * BC));
    dim3 blockSize(BN, BC);
    bev_pool_kernel_v2_outchannelfirst<float, float, float, TC, TN><<<gridSize, blockSize>>>(
        c, n_intervals, depth, feat, ranks_depth, ranks_feat,
        interval_starts, interval_lengths, out);
}


template<typename DType, typename FType, typename OType, int TC, int TN>
__global__ void bev_pool_kernel_v3shm(
    int C, int n_intervals,
    const DType *__restrict__ depth,
    const FType *__restrict__ feat,
    const int *__restrict__ ranks_depth,
    const int *__restrict__ ranks_feat,
    const int *__restrict__ ranks_bev,
    const int *__restrict__ interval_starts,
    const int *__restrict__ interval_lengths,
    OType *__restrict__ out) {

  extern __shared__ OType shout[8][32][TC];
  extern __shared__ int shvidx[8]; // voxel-id

  int tc_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int n_idx = blockIdx.y * blockDim.y + threadIdx.y;
  int n_idx_mod = n_idx % 8;
  float psum[TC];

  shvidx[n_idx_mod] = -1;
  for (int tc = 0; tc < TC; tc++) {
    psum[tc] = 0;
    shout[n_idx_mod][tc_idx][tc] = 0;
  }
  __syncthreads();


  int interval_start = __ldg(&interval_starts[n_idx]);
  int interval_length = __ldg(&interval_lengths[n_idx]);
  if (interval_start == -1) return;

  int v_idx = __ldg(&ranks_bev[interval_start]); // voxel index

  bool use_shmem = false;
  int v_idx_mod = v_idx % 8;
  int cas = atomicCAS(&shvidx[v_idx_mod], -1, v_idx);
  if (cas == -1 || cas == v_idx)
    use_shmem = true;
  __syncthreads();
    


  for (int i = 0; i < interval_length; i++) {
    float d = (float)__ldg(&depth[ranks_depth[interval_start + i]]);
#pragma unroll
    for (int tc = 0; tc < TC; tc++) {
      int c_idx = tc_idx * TC + tc;
      if (c_idx >= C) continue;

      float f = (float)__ldg(&feat[ranks_feat[interval_start + i] * C + c_idx]);
      psum[tc] = __fmaf_rn(d, f, psum[tc]);
    }
  }

#pragma unroll
  for (int tc = 0; tc < TC; tc++) {
    int c_idx = tc_idx * TC + tc;
    if (c_idx >= C) break;
    int bev_off;
    if (use_shmem) {
      atomicAdd(&shout[v_idx_mod][tc_idx][tc], psum[tc]);
    } else {
      bev_off = v_idx * C + c_idx;
      atomicAdd(&out[bev_off], psum[tc]);
    }
  }

  __syncthreads();
  for (int tc = 0; tc < TC; tc++) {
    int c_idx = tc_idx * TC + tc;
    if (c_idx >= C) break;
    if (shvidx[n_idx_mod] != -1) {
      int bev_off = shvidx[n_idx_mod] * C + c_idx;
      atomicAdd(&out[bev_off], shout[n_idx_mod][tc_idx][tc]);
    }
  }

}


template<typename DType, typename FType, typename OType, int TC, int TN>
__global__ void bev_pool_kernel_v3(
    int C, int n_intervals,
    const DType *__restrict__ depth,
    const FType *__restrict__ feat,
    const int *__restrict__ ranks_depth,
    const int *__restrict__ ranks_feat,
    //const int *__restrict__ ranks_bev,
    const int *__restrict__ interval_starts,
    const int *__restrict__ interval_lengths,
    const int *__restrict__ interval_vids,
    OType *__restrict__ out) {

  int tc_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int tn_idx = blockIdx.y * blockDim.y + threadIdx.y;

#pragma unroll
  for (int tn = 0; tn < TN; tn++) {
    float psum[TC];
    int n_idx = tn_idx * TN + tn;
    if (n_idx >= n_intervals) break;

    int interval_start = __ldg(&interval_starts[n_idx]);
    int interval_length = __ldg(&interval_lengths[n_idx]);

    if (interval_start == -1) break;
    //int v_idx = __ldg(&ranks_bev[interval_start]);
    int v_idx = __ldg(&interval_vids[n_idx]);

    for (int tc = 0; tc < TC; tc++) {
      psum[tc] = 0;
    }

    for (int i = 0; i < interval_length; i++) {
      float d = (float)__ldg(&depth[ranks_depth[interval_start + i]]);
#pragma unroll
      for (int tc = 0; tc < TC; tc++) {
        int c_idx = tc_idx * TC + tc;
        if (c_idx >= C) continue;

        float f = (float)__ldg(&feat[ranks_feat[interval_start + i] * C + c_idx]);
        psum[tc] = __fmaf_rn(d, f, psum[tc]);
      }
    }

#pragma unroll
    for (int tc = 0; tc < TC; tc++) {
      int c_idx = tc_idx * TC + tc;
      if (c_idx >= C) break;
      int bev_off = v_idx * C + c_idx;
      atomicAdd(&out[bev_off], psum[tc]);
    }
  }
}


extern "C"
void bev_pool_v3_float_float_float(int c, int n_intervals,
                                  const float *depth,
                                  const float *feat,
                                  const int *ranks_depth,
                                  const int *ranks_feat,
                                  //const int *ranks_bev,
                                  const int *interval_starts,
                                  const int *interval_lengths,
                                  const int *interval_vids,
                                  float *out) {

    printf("TC=%d, TN=%d, n_intervals=%d\n", TC, TN, n_intervals);
    dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
    dim3 blockSize(BC, BN);

    hipMemset(out, 0, OH*OW*C*sizeof(float));
    bev_pool_kernel_v3<float, float, float, TC, TN><<<gridSize, blockSize>>>(
        c, n_intervals, depth, feat, ranks_depth, ranks_feat, //ranks_bev,
        interval_starts, interval_lengths, interval_vids, out);
}


template<typename DType, typename FType, typename OType, const int TC, const int TN>
__global__ void bev_pool_kernel_v4(
    int C, int n_intervals,
    const DType *__restrict__ depth,
    const FType *__restrict__ feat,
    const int *__restrict__ ranks_depth,
    const int *__restrict__ ranks_feat,
    const int *__restrict__ interval_starts,
    const int *__restrict__ interval_lengths,
    const int *__restrict__ interval_vids,
    OType *__restrict__ out) {

  int tc_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int tn_idx = blockIdx.y * blockDim.y + threadIdx.y;

#pragma unroll
  for (int tn = 0; tn < TN; tn++) {
    float psum[TC];
    int n_idx = tn_idx * TN + tn;
    if (n_idx >= n_intervals) break;

    int interval_start = __ldg(&interval_starts[n_idx]);
    int interval_length = __ldg(&interval_lengths[n_idx]);
    int vid = __ldg(&interval_vids[n_idx]);

    if (interval_start == -1) break;

    for (int tc = 0; tc < TC; tc++) {
      psum[tc] = 0;
    }

    for (int i = 0; i < interval_length; i++) {
      float d = (float)__ldg(&depth[ranks_depth[interval_start + i]]);
#pragma unroll
      for (int tc = 0; tc < TC; tc++) {
        int c_idx = tc_idx * TC + tc;
        if (c_idx >= C) continue;

        float f = (float)__ldg(&feat[ranks_feat[interval_start + i] * C + c_idx]);
        psum[tc] = __fmaf_rn(d, f, psum[tc]);
      }
    }

#pragma unroll
    for (int tc = 0; tc < TC; tc++) {
      int c_idx = tc_idx * TC + tc;
      if (c_idx >= C) break;
      int bev_off = vid * C + c_idx;
      out[bev_off] = psum[tc];
    }
  }
}


extern "C"
void bev_pool_v4_float_float_float(int c, int n_intervals,
                                   const float *depth,
                                   const float *feat,
                                   const int *ranks_depth,
                                   const int *ranks_feat,
                                   const int *interval_starts,
                                   const int *interval_lengths,
                                   const int *interval_vids,
                                   float *out) {
    dim3 gridSize((c + TC * BC - 1)/(TC * BC), (n_intervals + TN * BN - 1)/(TN * BN));
    dim3 blockSize(BC, BN);
    bev_pool_kernel_v4<float, float, float, TC, TN><<<gridSize, blockSize>>>(
        c, n_intervals, depth, feat, ranks_depth, ranks_feat,
        interval_starts, interval_lengths, interval_vids, out);
}



int main() {
  auto gen_rand = get_rand_data_gen<float, std::uniform_real_distribution>(-10.f, 10.5f);

  size_t depth_element_count =  get_config("N") *
                                get_config("D") *
                                get_config("IH") *
                                get_config("IW");
  float *depth_float; CHECK(
    hipMallocManaged(
      &depth_float,
      depth_element_count * sizeof(float)
    )
  );
  for (int i = 0; i < depth_element_count; i++) depth_float[i] = gen_rand({});
  half *depth_half; CHECK(
    hipMallocManaged(
      &depth_half,
      depth_element_count * sizeof(half)
    )
  );
  for (int i = 0; i < depth_element_count; i++) depth_half[i] = gen_rand({});


  size_t feat_element_count = get_config("N") *
                              get_config("IH") *
                              get_config("IW") *
                              get_config("C");
  float *feat_float; CHECK(
    hipMallocManaged(
      &feat_float,
      feat_element_count * sizeof(float)
    )
  );
  for (int i = 0; i < feat_element_count; i++) feat_float[i] = gen_rand({});
  half *feat_half; CHECK(
    hipMallocManaged(
      &feat_half,
      feat_element_count * sizeof(half)
    )
  );
  for (int i = 0; i < feat_element_count; i++) feat_half[i] = gen_rand({});


  size_t out_element_count =  get_config("OH") *
                              get_config("OW") *
                              get_config("C");
  float *out_float; CHECK(
    hipMallocManaged(
      &out_float,
      out_element_count * sizeof(float)
    )
  );
  half *out_half; CHECK(
    hipMallocManaged(
      &out_half,
      out_element_count * sizeof(half)
    )
  );


  int *ranks_depth; CHECK(hipMallocManaged(&ranks_depth, 4000000 * sizeof(int)));
  int *ranks_feat; CHECK(hipMallocManaged(&ranks_feat, 4000000 * sizeof(int)));
  int *ranks_bev; CHECK(hipMallocManaged(&ranks_bev, 4000000 * sizeof(int)));
  int *interval_starts; CHECK(hipMallocManaged(&interval_starts, 50000 * sizeof(int)));
  int *interval_lengths; CHECK(hipMallocManaged(&interval_lengths, 50000 * sizeof(int)));
  int8_t *ranks_bev_mask; CHECK(hipMallocManaged(&ranks_bev_mask, OH * OW * sizeof(int8_t)));
  int *interval_starts_e; CHECK(hipMallocManaged(&interval_starts_e, 50000 * sizeof(int)));
  int *interval_lengths_e; CHECK(hipMallocManaged(&interval_lengths_e, 50000 * sizeof(int)));
  int *interval_vids_e; CHECK(hipMallocManaged(&interval_vids_e, 50000 * sizeof(int)));
  int *interval_starts_x; CHECK(hipMallocManaged(&interval_starts_x, 50000 * sizeof(int)));
  int *interval_lengths_x; CHECK(hipMallocManaged(&interval_lengths_x, 3000000 * sizeof(int)));
  int *interval_vids_x; CHECK(hipMallocManaged(&interval_vids_x, 3000000 * sizeof(int)));
  int *n_intervals_x; CHECK(hipMallocManaged(&n_intervals_x, 3000000 * sizeof(int)));

  tensor_init(ranks_depth,
              ranks_feat,
              ranks_bev,
              interval_starts,
              interval_lengths,
              ranks_bev_mask,
              interval_starts_e,
              interval_lengths_e,
              interval_vids_e,
              interval_starts_x,
              interval_lengths_x,
              interval_vids_x,
              n_intervals_x);

  bev_pool_float_float_float(
    get_config("C"),
    get_config("OH") * get_config("OW"),
    depth_float,
    feat_float,
    ranks_depth,
    ranks_feat,
    interval_starts_e,
    interval_lengths_e,
    out_float
  );


  bev_pool_float_float_float_nchw(
    get_config("C"),
    get_config("OH") * get_config("OW"),
    depth_float,
    feat_float,
    ranks_depth,
    ranks_feat,
    interval_starts_e,
    interval_lengths_e,
    out_float
  );

  bev_pool_half_float_half(
    get_config("C"),
    get_config("OH") * get_config("OW"),
    depth_half,
    feat_half,
    ranks_depth,
    ranks_feat,
    interval_starts_e,
    interval_lengths_e,
    out_half
  );

  bev_pool_half_float_float(
    get_config("C"),
    get_config("OH") * get_config("OW"),
    depth_half,
    feat_half,
    ranks_depth,
    ranks_feat,
    interval_starts_e,
    interval_lengths_e,
    out_float
  );

  bev_pool_half_half_half(
    get_config("C"),
    get_config("OH") * get_config("OW"),
    depth_half,
    feat_half,
    ranks_depth,
    ranks_feat,
    interval_starts_e,
    interval_lengths_e,
    out_half
  );

  bev_pool_v2_float_float_float(
    get_config("C"),
    get_config("OH") * get_config("OW"),
    depth_float,
    feat_float,
    ranks_depth,
    ranks_feat,
    interval_starts_e,
    interval_lengths_e,
    out_float
  );

  
  bev_pool_v2_float_float_float_outchannelfirst(
    get_config("C"),
    get_config("OH") * get_config("OW"),
    depth_float,
    feat_float,
    ranks_depth,
    ranks_feat,
    interval_starts_e,
    interval_lengths_e,
    out_float
  );

  bev_pool_v2_float_half_half(
    get_config("C"),
    get_config("OH") * get_config("OW"),
    depth_float,
    feat_half,
    ranks_depth,
    ranks_feat,
    interval_starts_e,
    interval_lengths_e,
    out_half
  );

  bev_pool_v2_float_half_float(
    get_config("C"),
    get_config("OH") * get_config("OW"),
    depth_float,
    feat_half,
    ranks_depth,
    ranks_feat,
    interval_starts_e,
    interval_lengths_e,
    out_float
  );

  bev_pool_v2_half_half_float(
    get_config("C"),
    get_config("OH") * get_config("OW"),
    depth_half,
    feat_half,
    ranks_depth,
    ranks_feat,
    interval_starts_e,
    interval_lengths_e,
    out_float
  );



  bev_pool_v2_half_float_float(
    get_config("C"),
    get_config("OH") * get_config("OW"),
    depth_half,
    feat_float,
    ranks_depth,
    ranks_feat,
    interval_starts_e,
    interval_lengths_e,
    out_float
  );



  bev_pool_v2_half_float_half(
    get_config("C"),
    get_config("OH") * get_config("OW"),
    depth_half,
    feat_float,
    ranks_depth,
    ranks_feat,
    interval_starts_e,
    interval_lengths_e,
    out_half
  );


  bev_pool_v3_float_float_float(
    get_config("C"),
    get_config("OH") * get_config("OW"),
    depth_float,
    feat_float,
    ranks_depth,
    ranks_feat,
    interval_starts_x,
    interval_lengths_x,
    interval_vids_x,
    out_float
  );


  CHECK(hipDeviceSynchronize());
}